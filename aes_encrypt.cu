#include <iostream>
#include <fstream>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

#define BLOCK_SIZE 16 // AES block size (128 bits)

// A simplified AES encryption step (just for demonstration; NOT actual AES)
__device__ void aes_encrypt_block(unsigned char* block, unsigned char* key) {
    for (int i = 0; i < BLOCK_SIZE; i++) {
        block[i] ^= key[i];  // Just an example, XOR with key (not AES encryption)
    }
}

// CUDA kernel for AES encryption
__global__ void AES_EncryptKernel(unsigned char* data, unsigned char* key, int numBlocks) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numBlocks) {
        unsigned char* block = &data[idx * BLOCK_SIZE];
        aes_encrypt_block(block, key);
    }
}

void encryptFile(const char* inputFile, const char* outputFile, const unsigned char* key) {
    ifstream ifs(inputFile, ios::binary | ios::ate);
    if (!ifs) {
        cerr << "Cannot open input file!" << endl;
        exit(1);
    }

    streampos fileSize = ifs.tellg();
    ifs.seekg(0, ios::beg);

    // Cast to size_t to avoid errors when calculating numBlocks
    size_t numBlocks = (static_cast<size_t>(fileSize) + BLOCK_SIZE - 1) / BLOCK_SIZE; // Round up number of blocks
    unsigned char* data = new unsigned char[numBlocks * BLOCK_SIZE];
    ifs.read(reinterpret_cast<char*>(data), fileSize);
    ifs.close();

    // Allocate device memory for data and key
    unsigned char* d_data;
    unsigned char* d_key;
    hipMalloc(&d_data, numBlocks * BLOCK_SIZE);
    hipMalloc(&d_key, BLOCK_SIZE);

    // Copy data to device memory
    hipMemcpy(d_data, data, numBlocks * BLOCK_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, BLOCK_SIZE, hipMemcpyHostToDevice);

    // Launch the encryption kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numBlocks + threadsPerBlock - 1) / threadsPerBlock;
    AES_EncryptKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_key, numBlocks);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << endl;
        exit(1);
    }

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy the result back to host memory
    hipMemcpy(data, d_data, numBlocks * BLOCK_SIZE, hipMemcpyDeviceToHost);

    // Write the encrypted data to the output file
    ofstream ofs(outputFile, ios::binary);
    ofs.write(reinterpret_cast<char*>(data), numBlocks * BLOCK_SIZE);
    ofs.close();

    // Free device memory
    hipFree(d_data);
    hipFree(d_key);

    // Free host memory
    delete[] data;
}

int main() {
    // AES key (128 bits)
    unsigned char key[16] = {0x2b, 0x7e, 0x15, 0x16, 0x28, 0xae, 0xd2, 0xa6,
                             0xab, 0xf7, 0x97, 0x75, 0x46, 0x38, 0x6d, 0x60};

    const char* inputFile = "./novel.txt";   // Path to your input file
    const char* encryptedFile = "./encrypted.bin"; // Output encrypted file

    // Encrypt the file
    encryptFile(inputFile, encryptedFile, key);
    cout << "File encrypted successfully!" << endl;

    return 0;
}
